#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <cuda/cuda_runtime.h>


using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__constant__ long int filter[5000];

__global__ void convolution_kernel(long int *matrix, long int *result, int h, int w, int c, int r, int s, int k)
{
    extern __shared__ long int SM[];
    long int *s_matrix = SM;
    long int *s_result = SM + w;

    int filter_row = blockIdx.z;
    int filter_Number = blockIdx.y/c;
    int channel_number = blockIdx.y%c;
    int fid = r*s*c*filter_Number + r*s*channel_number + s*filter_row;

    int col = threadIdx.x;
    int row = blockIdx.x + filter_row - r/2;

    int m = channel_number*h + row;
    int n = col - s/2;

    s_matrix[col] = (row>=0 && row<h) ? matrix[m*w + col] : 0;
    s_result[col] = 0;

    __syncthreads();

    int rid = h*w*filter_Number + w*blockIdx.x + col;

    for(int i=0; i<s; ++i){
      if(n+i>=0 && n+i<w && col<w)
        s_result[col] += s_matrix[n+i]*filter[fid + i];
    }
    
    atomicAdd((unsigned long long int*)&result[rid], (unsigned long long int)s_result[col]);
}

int main(int argc, char **argv)
{
    int h, w, c;
    cin >> h >> w >> c;
    long int *h_mat = new long int[h * w * c];
    for (long int i = 0; i < h * w * c; i++)
    {
        cin >> h_mat[i];
    }

    int cf, r, s, k;
    cin >> cf >> r >> s >> k;

    long int *h_filter = new long int[r * s * c * k];
    for (long int i = 0; i < r * s * c * k; i++)
    {
        cin >> h_filter[i];
    }
    long int *h_ans = new long int[h * w * k];


    auto start = std::chrono::high_resolution_clock::now(); 

    long int *matrix, *result;

    hipMalloc(&matrix, w*h*c*sizeof(long int));
    hipMalloc(&result, h*w*k*sizeof(long int));

    hipMemcpy(matrix, h_mat, w*h*c*sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(filter), h_filter, r*s*c*k*sizeof(long int));
    hipMemset(result, 0, h*w*k*sizeof(long int));

    dim3 grid(h, k*c, r);

    convolution_kernel<<<grid, w, (2*w)*sizeof(long int)>>>(matrix, result, h, w, c, r, s, k);

    hipMemcpy(h_ans, result, h*w*k*sizeof(long int), hipMemcpyDeviceToHost);

    hipFree(matrix);
    hipFree(result);

    auto end = std::chrono::high_resolution_clock::now(); 
    std::chrono::duration<double> elapsed1 = end - start;


    hipDeviceSynchronize();
    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < h * k; i++)
        {
            for (long int j = 0; j < w; j++)
            {
                file << h_ans[i * w + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}
